#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <stdint.h>

#include "cuda_compat.h"

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

#include "quantization/fp8/quant_utils.cuh"

namespace vllm {

template<typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void reshape_and_cache_kernel(
  const scalar_t* __restrict__ key,           // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ value,         // [num_tokens, num_heads, head_size]
  cache_t* __restrict__ key_cache,           // [num_blocks, num_heads, head_size/x, block_size, x]
  cache_t* __restrict__ value_cache,         // [num_blocks, num_heads, head_size, block_size]
  const int64_t* __restrict__ slot_mapping,   // [num_tokens]
  const int key_stride,
  const int value_stride,
  const int num_heads,
  const int head_size,
  const int block_size,
  const float *key_scale,
  const float *value_scale,
  const int x) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  if (slot_idx < 0) {
    // Padding token that should be ignored.
    return;
  }

  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;

  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t src_key_idx = token_idx * key_stride + i;
    const int64_t src_value_idx = token_idx * value_stride + i;

    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int x_idx = head_offset / x;
    const int x_offset = head_offset % x;

    const int64_t tgt_key_idx = block_idx * num_heads * (head_size / x) * block_size * x
                                + head_idx * (head_size / x) * block_size * x
                                + x_idx * block_size * x
                                + block_offset * x
                                + x_offset;
    const int64_t tgt_value_idx = block_idx * num_heads * head_size * block_size
                                  + head_idx * head_size * block_size
                                  + head_offset * block_size
                                  + block_offset;
    scalar_t tgt_key = key[src_key_idx];
    scalar_t tgt_value = value[src_value_idx];
    if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
      key_cache[tgt_key_idx] = tgt_key;
      value_cache[tgt_value_idx] = tgt_value;
    } else {
      key_cache[tgt_key_idx] =
          fp8::scaled_convert<cache_t, scalar_t, kv_dt>(tgt_key, *key_scale);
      value_cache[tgt_value_idx] =
          fp8::scaled_convert<cache_t, scalar_t, kv_dt>(tgt_value, *value_scale);
    }
  }
}

//  CACHE_T is the stored data type of kv-cache.
// KV_T is the data type of key and value tensors.
// KV_DTYPE is the real data type of kv-cache.
#define CALL_RESHAPE_AND_CACHE(KV_T, CACHE_T, KV_DTYPE)               \
  vllm::reshape_and_cache_kernel<KV_T, CACHE_T, KV_DTYPE><<<grid, block, 0, stream>>>(      \
    reinterpret_cast<KV_T*>(key),                                        \
    reinterpret_cast<KV_T*>(value),                                      \
    reinterpret_cast<CACHE_T*>(key_cache),                                  \
    reinterpret_cast<CACHE_T*>(value_cache),                                \
    slot_mapping,                                                     \
    key_stride,                                                       \
    value_stride,                                                     \
    num_heads,                                                        \
    head_size,                                                        \
    block_size,                                                       \
    key_scale,                                                        \
    value_scale,                                                      \
    x);


} // namespace vllm

extern "C" void reshape_and_cache(
  void *key,              // [num_tokens, num_heads, head_size]
  void *value,            // [num_tokens, num_heads, head_size]
  void *key_cache,        // [num_blocks, num_heads, head_size/x, block_size, x]
  void *value_cache,      // [num_blocks, num_heads, head_size, block_size]
  int64_t* slot_mapping,  // [num_tokens]

  int32_t num_tokens,
  int32_t num_heads,
  int32_t head_size,
  int32_t block_size,
  int32_t x,
  int32_t key_stride,
  int32_t value_stride,
  
  const float *key_scale,
  const float *value_scale,

  uint32_t dtype,      // 0 => f16; 1 => bf16; 2 => f32
  uint32_t kv_dtype      // 0 => same as dtype; 1 => fp8e4m3
  )
{
  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));
  const hipStream_t stream = 0;

  DISPATCH_BY_KV_CACHE_DTYPE(dtype, kv_dtype, CALL_RESHAPE_AND_CACHE)
}